#include "hip/hip_runtime.h"
﻿// smallptCUDA by Sam Lapere, 2015
// based on smallpt, a path tracer by Kevin Beason, 2008  

#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include "hip/hip_vector_types.h"// from http://www.icmc.usp.br/~castelo/CUDA/common/inc/cutil_math.h
#include <hip/hip_runtime.h>
#include <GL/glew.h>
#include <GL/freeglut.h>
#define __CUDA_INTERNAL_COMPILATION__
#include "math_functions.h"
#undef __CUDA_INTERNAL_COMPILATION__
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include ""

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>



#include "cuda_render.h"
#include "linear_algebra.h"
 float3* accumulatebuffer;
 unsigned int vbo;
 float* dev_triangle_p; // the cuda device pointer that points to the uploaded scene_triangles
// output buffer
 float3* dptr;
#define BVH_STACK_SIZE 8
int total_number_of_scene_triangles = 0;


// scene bounding box
float3 scene_aabbox_min;
float3 scene_aabbox_max;

// the scene scene_triangles are stored in a 1D CUDA texture of float4 for memory alignment
// store two edges instead of vertices
// each triangle is stored as three float4s: (float4 first_vertex, float4 edge1, float4 edge2)
texture<float4, 1, hipReadModeElementType> Texture_triangle;
texture<float4, 1, hipReadModeElementType> bvhNodesPosTexture;
texture<uint4, 1, hipReadModeElementType> Texture_tree_info;
texture<float2, 1, hipReadModeElementType> Texture_bvh_slab; // 用于检测bvh的包围盒

// hardcoded camera position
__device__ float3 firstcamorig = { 50, 52, 295.6 };

// OpenGL vertex buffer object for real-time viewport

void* d_vbo_buffer = NULL;

__device__ __inline__ int   min_min(int a, int b, int c) { int v; asm("vmin.s32.s32.s32.min %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ int   min_max(int a, int b, int c) { int v; asm("vmin.s32.s32.s32.max %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ int   max_min(int a, int b, int c) { int v; asm("vmax.s32.s32.s32.min %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ int   max_max(int a, int b, int c) { int v; asm("vmax.s32.s32.s32.max %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ float fmin_fmin(float a, float b, float c) { return __int_as_float(min_min(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmin_fmax(float a, float b, float c) { return __int_as_float(min_max(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmax_fmin(float a, float b, float c) { return __int_as_float(max_min(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmax_fmax(float a, float b, float c) { return __int_as_float(max_max(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }

__device__ __inline__ float spanBeginKepler(float a0, float a1, float b0, float b1, float c0, float c1, float d) { return fmax_fmax(fminf(a0, a1), fminf(b0, b1), fmin_fmax(c0, c1, d)); }
__device__ __inline__ float spanEndKepler(float a0, float a1, float b0, float b1, float c0, float c1, float d) { return fmin_fmin(fmaxf(a0, a1), fmaxf(b0, b1), fmax_fmin(c0, c1, d)); }

// standard ray box intersection routines (for debugging purposes only)
// based on Intersect::RayBox() in original Aila/Laine code
__device__ __inline__ float spanBeginKepler2(float lo_x, float hi_x, float lo_y, float hi_y, float lo_z, float hi_z, float d) {

	Vector3f t0 = Vector3f(lo_x, lo_y, lo_z);
	Vector3f t1 = Vector3f(hi_x, hi_y, hi_z);

	Vector3f realmin = min3f(t0, t1);

	float raybox_tmin = realmin.max(); // maxmin

	//return Vec2f(tmin, tmax);
	return raybox_tmin;
}

__device__ __inline__ float spanEndKepler2(float lo_x, float hi_x, float lo_y, float hi_y, float lo_z, float hi_z, float d) {

	Vector3f t0 = Vector3f(lo_x, lo_y, lo_z);
	Vector3f t1 = Vector3f(hi_x, hi_y, hi_z);

	Vector3f realmax = max3f(t0, t1);

	float raybox_tmax = realmax.min(); /// minmax

	//return Vec2f(tmin, tmax);
	return raybox_tmax;
}


__device__ Vector3f intersectRayTriangle(const Vector4f& rayOri, const Vector4f& rayDir, const Vector3f& v0, const Vector3f& v1, const Vector3f& v2)
{
	const float EPSILON = 0.00001f;
	const Vector3f miss(WORLD_MAX, WORLD_MAX, WORLD_MAX);

	Vector3f edge_u = v1 - v0;
	Vector3f edge_v = v2 - v0;

	Vector3f vec_t = Vector3f(rayOri.x, rayOri.y, rayOri.z) - v0;
	Vector3f vec_p = cross(Vector3f(rayDir.x, rayDir.y, rayDir.z), edge_v);
	float det = dot(edge_u, vec_p);
	float invdet = 1.0f / det;
	float u = dot(vec_t, vec_p) * invdet;
	Vector3f vec_q = cross(vec_t, edge_u);
	float v = dot(Vector3f(rayDir.x, rayDir.y, rayDir.z), edge_u);

	if (det > EPSILON)
	{
		if (u < 0.0f || u > 1.0f) return miss; // 1.0 want = det * 1/det  
		if (v < 0.0f || (u + v) > 1.0f) return miss;
		// if u and v are within these bounds, continue and go to float t = dot(...	           
	}

	else if (det < -EPSILON)
	{
		if (u > 0.0f || u < 1.0f) return miss;
		if (v > 0.0f || (u + v) < 1.0f) return miss;
		// else continue
	}

	else // if det is not larger (more positive) than EPSILON or not smaller (more negative) than -EPSILON, there is a "miss"
		return miss;

	float t = dot(edge_v, vec_q) * invdet;

	if (t > rayOri.w && t < rayOri.w)
		return Vector3f(u, v, t);

	// otherwise (t < raytmin or t > raytmax) miss
	return miss;
}

struct Ray {
	float3 orig;	// ray origin
	float3 dir;		// ray direction	
	__device__ Ray(float3 o_, float3 d_) : orig(o_), dir(d_) {}
};

enum Refl_t { DIFF, SPEC, REFR };  // material types, used in radiance(), only DIFF used here

// SPHERES

struct Sphere {

	float rad;				// radius 
	float3 pos, emi, col;	// position, emission, color 
	Refl_t refl;			// reflection type (DIFFuse, SPECular, REFRactive)

	__device__ float intersect(const Ray& r) const { // returns distance, 0 if nohit 

		// Ray/sphere intersection
		// Quadratic formula required to solve ax^2 + bx + c = 0 
		// Solution x = (-b +- sqrt(b*b - 4ac)) / 2a
		// Solve t^2*d.d + 2*t*(o-p).d + (o-p).(o-p)-R^2 = 0 

		float3 op = pos - r.orig;  // 
		float t, epsilon = 0.01f;
		float b = dot(op, r.dir);
		float disc = b * b - dot(op, op) + rad * rad; // discriminant
		if (disc < 0) return 0; else disc = sqrtf(disc);
		return (t = b - disc) > epsilon ? t : ((t = b + disc) > epsilon ? t : 0);
	}
};

__device__ bool RayBoxIntersection_bool(int boxIdx, const float3& rayOri, const float3& rayDir)
{
	float slab_near = -WORLD_MAX;
	float slab_far = WORLD_MAX;

	float2 slab_intersect = tex1Dfetch(Texture_bvh_slab, 3 * boxIdx);

	if (rayDir.x == 0.0f)
	{
		if (rayOri.x < slab_intersect.x)return false;
		if (rayOri.x > slab_intersect.y)return false;
	}
	else
	{
		float t1 = (slab_intersect.x - rayOri.x) / rayDir.x;
		float t2 = (slab_intersect.y - rayOri.x) / rayDir.x;
		if (t1 > t2)
		{
			float temp = t1;
			t1 = t2;
			t2 = temp;
		}
		if (t1 > slab_near)slab_near = t1;
		if (t2 > slab_far)slab_far = t2;
		if (slab_near > slab_far) return false;
		if (slab_far < 0.0f)return false;
	}
	slab_intersect = tex1Dfetch(Texture_bvh_slab, 3 * boxIdx + 1);

	if (rayDir.y == 0.0f)
	{
		if (rayOri.y < slab_intersect.x)return false;
		if (rayOri.y > slab_intersect.y)return false;
	}
	else
	{
		float t1 = (slab_intersect.x - rayOri.y) / rayDir.y;
		float t2 = (slab_intersect.y - rayOri.y) / rayDir.y;
		if (t1 > t2)
		{
			float temp = t1;
			t1 = t2;
			t2 = temp;
		}
		if (t1 > slab_near)slab_near = t1;
		if (t2 > slab_far)slab_far = t2;
		if (slab_near > slab_far) return false;
		if (slab_far < 0.0f)return false;
	}
	slab_intersect = tex1Dfetch(Texture_bvh_slab, 3 * boxIdx + 2);

	if (rayDir.y == 0.0f)
	{
		if (rayOri.y < slab_intersect.x)return false;
		if (rayOri.y > slab_intersect.y)return false;
	}
	else
	{
		float t1 = (slab_intersect.x - rayOri.y) / rayDir.y;
		float t2 = (slab_intersect.y - rayOri.y) / rayDir.y;
		if (t1 > t2)
		{
			float temp = t1;
			t1 = t2;
			t2 = temp;
		}
		if (t1 > slab_near)slab_near = t1;
		if (t2 > slab_far)slab_far = t2;
		if (slab_near > slab_far) return false;
		if (slab_far < 0.0f)return false;
	}
	return true;
}

// scene_triangles

// the classic ray triangle intersection: http://www.cs.virginia.edu/~gfx/Courses/2003/ImageSynthesis/papers/Acceleration/Fast%20MinimumStorage%20RayTriangle%20Intersection.pdf
// for an explanation see http://www.scratchapixel.com/lessons/3d-basic-rendering/ray-tracing-rendering-a-triangle/moller-trumbore-ray-triangle-intersection

__device__ float RayTriangleIntersection(const Ray& r,
	const float3& v0,
	const float3& edge1,
	const float3& edge2)
{

	float3 tvec = r.orig - v0;
	float3 pvec = cross(r.dir, edge2);
	float  det = dot(edge1, pvec);

	det = __fdividef(1.0f, det);  // CUDA intrinsic function 

	float u = dot(tvec, pvec) * det;

	if (u < 0.0f || u > 1.0f)
		return -1.0f;

	float3 qvec = cross(tvec, edge1);

	float v = dot(r.dir, qvec) * det;

	if (v < 0.0f || (u + v) > 1.0f)
		return -1.0f;

	return dot(edge2, qvec) * det;
}


__device__ float3 getTriangleNormal(const int triangleIndex) {

	float4 edge1 = tex1Dfetch(Texture_triangle, triangleIndex * 3 + 1);
	float4 edge2 = tex1Dfetch(Texture_triangle, triangleIndex * 3 + 2);

	// cross product of two triangle edges yields a vector orthogonal to triangle plane
	float3 trinormal = cross(make_float3(edge1.x, edge1.y, edge1.z), make_float3(edge2.x, edge2.y, edge2.z));
	trinormal = normalize(trinormal);

	return trinormal;
}

__device__ void intersect_bvh(const Ray& r, float& t_scene, int& triangle_id, int& geomtype)
{
	int stack[BVH_STACK_SIZE];
	int stackIdx = 0;
	stack[stackIdx++] = 0;
	while (stackIdx > 0)
	{
		int boxIdx = stack[stackIdx - 1];
		uint4 node_info = tex1Dfetch(Texture_tree_info, boxIdx);
		stackIdx -= 1;
		if (node_info.x)
		{
			// is leaf
			int tri_idx = node_info.w;
			float4 v0 = tex1Dfetch(Texture_triangle, tri_idx * 3 + 0);
			float4 edge1 = tex1Dfetch(Texture_triangle, tri_idx * 3 + 1);
			float4 edge2 = tex1Dfetch(Texture_triangle, tri_idx * 3 + 2);

			// intersect ray with reconstructed triangle	
			float t = RayTriangleIntersection(r,
				make_float3(v0.x, v0.y, v0.z),
				make_float3(edge1.x, edge1.y, edge1.z),
				make_float3(edge2.x, edge2.y, edge2.z));

			// keep track of closest distance and closest triangle
			// if ray/tri intersection finds an intersection point that is closer than closest intersection found so far
			if (t < t_scene && t > 0.001)
			{
				t_scene = t;
				triangle_id = tri_idx;
				geomtype = 3;
			}

		}
		else
		{
			
			if (RayBoxIntersection_bool(boxIdx, r.orig, r.dir))
			{
				stack[stackIdx++] = node_info.y;
				stack[stackIdx++] = node_info.z;
			}
			if (stackIdx > BVH_STACK_SIZE)
			{
				return;
			}
		}

	}
}

__device__ void intersectAllscene_triangles(const Ray& r, float& t_scene, int& triangle_id, const int number_of_scene_triangles, int& geomtype) {

	
	for (int i = 0; i < number_of_scene_triangles; i++)
	{
		// the scene_triangles are packed into the 1D texture using three consecutive float4 structs for each triangle, 
		// first float4 contains the first vertex, second float4 contains the first precomputed edge, third float4 contains second precomputed edge like this: 
		// (float4(vertex.x,vertex.y,vertex.z, 0), float4 (egde1.x,egde1.y,egde1.z,0),float4 (egde2.x,egde2.y,egde2.z,0)) 

		// i is triangle index, each triangle represented by 3 float4s in Texture_triangle
		float4 v0 = tex1Dfetch(Texture_triangle, i * 3);
		float4 edge1 = tex1Dfetch(Texture_triangle, i * 3 + 1);
		float4 edge2 = tex1Dfetch(Texture_triangle, i * 3 + 2);

		// intersect ray with reconstructed triangle	
		float t = RayTriangleIntersection(r,
			make_float3(v0.x, v0.y, v0.z),
			make_float3(edge1.x, edge1.y, edge1.z),
			make_float3(edge2.x, edge2.y, edge2.z));

		// keep track of closest distance and closest triangle
		// if ray/tri intersection finds an intersection point that is closer than closest intersection found so far
		if (t < t_scene && t > 0.001)
		{
			t_scene = t;
			triangle_id = i;
			geomtype = 3;
		}
	}
}
__device__ void intersectBVH(const float4 rayOri, const float4 rayDir)
{
	int thread_index = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
}

// AXIS ALIGNED BOXES

// helper functions
inline __device__ float3 minf3(float3 a, float3 b) { return make_float3(a.x < b.x ? a.x : b.x, a.y < b.y ? a.y : b.y, a.z < b.z ? a.z : b.z); }
inline __device__ float3 maxf3(float3 a, float3 b) { return make_float3(a.x > b.x ? a.x : b.x, a.y > b.y ? a.y : b.y, a.z > b.z ? a.z : b.z); }
inline __device__ float minf1(float a, float b) { return a < b ? a : b; }
inline __device__ float maxf1(float a, float b) { return a > b ? a : b; }

struct Box {

	float3 min; // minimum bounds
	float3 max; // maximum bounds
	float3 emi; // emission
	float3 col; // colour
	Refl_t refl; // material type

	// ray/box intersection
	// for theoretical background of the algorithm see 
	// http://www.scratchapixel.com/lessons/3d-basic-rendering/minimal-ray-tracer-rendering-simple-shapes/ray-box-intersection
	// optimised code from http://www.gamedev.net/topic/495636-raybox-collision-intersection-point/
	__device__ float intersect(const Ray& r) const {

		float epsilon = 0.001f; // required to prevent self intersection

		float3 tmin = (min - r.orig) / r.dir;
		float3 tmax = (max - r.orig) / r.dir;

		float3 real_min = minf3(tmin, tmax);
		float3 real_max = maxf3(tmin, tmax);

		float minmax = minf1(minf1(real_max.x, real_max.y), real_max.z); // 射线离开boundingbox后，与另一个轴的交点这段路程
		float maxmin = maxf1(maxf1(real_min.x, real_min.y), real_min.z); // 射线与一个轴相交后，到与boundingbox相交这段前的路程

		if (minmax >= maxmin) { return maxmin > epsilon ? maxmin : 0; }
		else return 0;
	}

	// calculate normal for point on axis aligned box
	__device__ float3 Box::normalAt(float3& point) {

		float3 normal = make_float3(0.f, 0.f, 0.f);
		float min_distance = 1e8;
		float distance;
		float epsilon = 0.001f;

		if (fabs(min.x - point.x) < epsilon) normal = make_float3(-1, 0, 0);
		else if (fabs(max.x - point.x) < epsilon) normal = make_float3(1, 0, 0);
		else if (fabs(min.y - point.y) < epsilon) normal = make_float3(0, -1, 0);
		else if (fabs(max.y - point.y) < epsilon) normal = make_float3(0, 1, 0);
		else if (fabs(min.z - point.z) < epsilon) normal = make_float3(0, 0, -1);
		else normal = make_float3(0, 0, 1);

		return normal;
	}
};

// scene: 9 spheres forming a Cornell box
// small enough to fit in constant GPU memory
__constant__ Sphere spheres[] = {
	// FORMAT: { float radius, float3 position, float3 emission, float3 colour, Refl_t material }
	// cornell box
	//{ 1e5f, { 1e5f + 1.0f, 40.8f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { 0.75f, 0.25f, 0.25f }, DIFF }, //Left 1e5f
	//{ 1e5f, { -1e5f + 99.0f, 40.8f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .25f, .25f, .75f }, DIFF }, //Right 
	//{ 1e5f, { 50.0f, 40.8f, 1e5f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF }, //Back 
	//{ 1e5f, { 50.0f, 40.8f, -1e5f + 600.0f }, { 0.0f, 0.0f, 0.0f }, { 0.00f, 0.00f, 0.00f }, DIFF }, //Front 
	//{ 1e5f, { 50.0f, -1e5f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF }, //Bottom 
	//{ 1e5f, { 50.0f, -1e5f + 81.6f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF }, //Top 
	//{ 16.5f, { 27.0f, 16.5f, 47.0f }, { 0.0f, 0.0f, 0.0f }, { 0.99f, 0.99f, 0.99f }, SPEC }, // small sphere 1
	//{ 16.5f, { 73.0f, 16.5f, 78.0f }, { 0.0f, 0.f, .0f }, { 0.09f, 0.49f, 0.3f }, REFR }, // small sphere 2
	//{ 600.0f, { 50.0f, 681.6f - .5f, 81.6f }, { 3.0f, 2.5f, 2.0f }, { 0.0f, 0.0f, 0.0f }, DIFF }  // Light 12, 10 ,8

	//outdoor scene: radius, position, emission, color, material

	//{ 1600, { 3000.0f, 10, 6000 }, { 37, 34, 30 }, { 0.f, 0.f, 0.f }, DIFF },  // 37, 34, 30 // sun
	//{ 1560, { 3500.0f, 0, 7000 }, { 50, 25, 2.5 }, { 0.f, 0.f, 0.f }, DIFF },  //  150, 75, 7.5 // sun 2
	
	{ 10000, { 50.0f, 40.8f, -1060 }, { 0.0003, 0.01, 0.15 }, { 0.175f, 0.175f, 0.25f }, DIFF }, // sky
	{ 100000, { 50.0f, -100000, 0 }, { 0.0, 0.0, 0 }, { 0.8f, 0.2f, 0.f }, DIFF }, // ground
	{ 82.5, { 30.0f, 180.5, 42 }, { 16, 12, 6 }, { .6f, .6f, 0.6f }, DIFF },  // small sphere 1
	/*
	{ 110000, { 50.0f, -110048.5, 0 }, { 3.6, 2.0, 0.2 }, { 0.f, 0.f, 0.f }, DIFF },  // horizon brightener
	
	{ 4e4, { 50.0f, -4e4 - 30, -3000 }, { 0, 0, 0 }, { 0.2f, 0.2f, 0.2f }, DIFF }, // mountains
	
	
	{ 12, { 115.0f, 10, 105 }, { 0.0, 0.0, 0.0 }, { 0.9f, 0.9f, 0.9f }, REFR },  // small sphere 2
	{ 22, { 65.0f, 22, 24 }, { 0, 0, 0 }, { 0.9f, 0.9f, 0.9f }, SPEC }, // small sphere 3
	
	*/
};

__constant__ Box boxes[] = {
	// FORMAT: { float3 minbounds,    float3 maxbounds,         float3 emission,    float3 colour,       Refl_t }
	
	/*{ { 5.0f, 0.0f, 70.0f }, { 45.0f, 11.0f, 115.0f }, { .0f, .0f, 0.0f }, { 0.5f, 0.5f, 0.5f }, DIFF },
	{ { 85.0f, 0.0f, 95.0f }, { 95.0f, 20.0f, 105.0f }, { .0f, .0f, 0.0f }, { 0.5f, 0.5f, 0.5f }, DIFF },*/
	{ { 75.0f, 20.0f, 85.0f }, { 105.0f, 22.0f, 115.0f }, { .0f, .0f, 0.0f }, { 0.5f, 0.5f, 0.5f }, DIFF },
	
};


__device__ inline bool intersect_scene(const Ray& r, float& t, int& sphere_id, int& box_id, int& triangle_id, const int number_of_scene_triangles, int& geomtype, const float3& bbmin, const float3& bbmax) {

	float tmin = 1e20;
	float tmax = -1e20;
	float d = 1e21;
	float k = 1e21;
	float q = 1e21;
	float inf = t = 1e20;
	
	// SPHERES
	// intersect all spheres in the scene
	float numspheres = sizeof(spheres) / sizeof(Sphere);
	for (int i = int(numspheres); i--;)  // for all spheres in scene
		// keep track of distance from origin to closest intersection point
		if ((d = spheres[i].intersect(r)) && d < t) { t = d; sphere_id = i; geomtype = 1; }

	// BOXES
	// intersect all boxes in the scene
	float numboxes = sizeof(boxes) / sizeof(Box);
	for (int i = int(numboxes); i--;) // for all boxes in scene
		if ((k = boxes[i].intersect(r)) && k < t) { t = k; box_id = i; geomtype = 2; }
	
	// scene_triangles
	Box scene_bbox; // bounding box around triangle meshes
	scene_bbox.min = bbmin;
	scene_bbox.max = bbmax;

	// if ray hits bounding box of triangle meshes, intersect ray with all scene_triangles
    //scene_bbox.intersect(r)
	intersect_bvh(r, t, triangle_id, geomtype);
	//intersectAllscene_triangles(r, t, triangle_id, number_of_scene_triangles, geomtype);

	// t is distance to closest intersection of ray with all primitives in the scene (spheres, boxes and scene_triangles)
	return t < inf;
}




__device__ void intersecctBVHTriangle()
{
	int thread_index = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	
	int nodeAddress = 0; // 从 0 号点开始
	
	while (true)
	{
		while (true)
		{
			float4 child_pos_x = tex1Dfetch(bvhNodesPosTexture, nodeAddress);
			float4 child_pos_y = tex1Dfetch(bvhNodesPosTexture, nodeAddress);
			float4 child_pos_z = tex1Dfetch(bvhNodesPosTexture, nodeAddress);
		}
	}
}

// radiance function
// compute path bounces in scene and accumulate returned color from each path sgment
__device__ float3 radiance(Ray& r, hiprandState* randstate, const int totaltris, const float3& scene_aabb_min, const float3& scene_aabb_max) { // returns ray color

	// colour mask
	float3 mask = make_float3(1.0f, 1.0f, 1.0f);
	// accumulated colour
	float3 accucolor = make_float3(0.0f, 0.0f, 0.0f);

	for (int bounces = 0; bounces < 5; bounces++) {  // iteration up to 4 bounces (instead of recursion in CPU code)

		// reset scene intersection function parameters
		float t = 100000; // distance to intersection 
		int sphere_id = -1;
		int box_id = -1;   // index of intersected sphere 
		int triangle_id = -1;
		int geomtype = -1;
		float3 f;  // primitive colour
		float3 emit; // primitive emission colour
		float3 x; // intersection point
		float3 n; // normal
		float3 nl; // oriented normal
		float3 d; // ray direction of next path segment
		Refl_t refltype;

		int hitSphereIdx = -1;
		int hitTriIdx = -1;
		int bestTriIdx = -1;
		float hitSphereDist = 1e20;
		float hitDistance = 1e20;

		// intersect ray with scene
		// intersect_scene keeps track of closest intersected primitive and distance to closest intersection point
		if (!intersect_scene(r, t, sphere_id, box_id, triangle_id, totaltris, geomtype, scene_aabb_min, scene_aabb_max))
			return make_float3(0.0f, 0.0f, 0.0f); // if miss, return black

		// else: we've got a hit with a scene primitive
		// determine geometry type of primitive: sphere/box/triangle

		// if sphere:
		if (geomtype == 1) {
			Sphere& sphere = spheres[sphere_id]; // hit object with closest intersection
			x = r.orig + r.dir * t;  // intersection point on object
			n = normalize(x - sphere.pos);		// normal
			nl = dot(n, r.dir) < 0 ? n : n * -1; // correctly oriented normal
			f = sphere.col;   // object colour
			refltype = sphere.refl;
			emit = sphere.emi;  // object emission
			accucolor += (mask * emit);
		}

		// if box:
		if (geomtype == 2) {
			Box& box = boxes[box_id];
			x = r.orig + r.dir * t;  // intersection point on object
			n = normalize(box.normalAt(x)); // normal
			nl = dot(n, r.dir) < 0 ? n : n * -1;  // correctly oriented normal
			f = box.col;  // box colour
			refltype = box.refl;
			emit = box.emi; // box emission
			accucolor += (mask * emit);
		}

		// if triangle:
		if (geomtype == 3) {
			int tri_index = triangle_id;
			x = r.orig + r.dir * t;  // intersection point
			n = normalize(getTriangleNormal(tri_index));  // normal 
			nl = dot(n, r.dir) < 0 ? n : n * -1;  // correctly oriented normal

			// colour, refltype and emit value are hardcoded and apply to all scene_triangles
			// no per triangle material support yet
			f = make_float3(0.9f, 0.4f, 0.1f);  // triangle colour
			refltype = DIFF;
			emit = make_float3(0.0f, 0.0f, 0.0f);
			accucolor += (mask * emit);
		}

		// SHADING: diffuse, specular or refractive

		// ideal diffuse reflection (see "Realistic Ray Tracing", P. Shirley)
		if (refltype == DIFF) {

			// create 2 random numbers
			float r1 = 2 * M_PI * hiprand_uniform(randstate);
			float r2 = hiprand_uniform(randstate);
			float r2s = sqrtf(r2);

			// compute orthonormal coordinate frame uvw with hitpoint as origin 
			float3 w = nl;
			float3 u = normalize(cross((fabs(w.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), w));
			float3 v = cross(w, u);

			// compute cosine weighted random ray direction on hemisphere 
			d = normalize(u * cos(r1) * r2s + v * sin(r1) * r2s + w * sqrtf(1 - r2));

			// offset origin next path segment to prevent self intersection
			x += nl * 0.03;

			// multiply mask with colour of object
			mask *= f;
		}

		// ideal specular reflection (mirror) 
		if (refltype == SPEC) {

			// compute relfected ray direction according to Snell's law
			d = r.dir - 2.0f * n * dot(n, r.dir);

			// offset origin next path segment to prevent self intersection
			x += nl * 0.01f;

			// multiply mask with colour of object
			mask *= f;
		}

		// ideal refraction (based on smallpt code by Kevin Beason)
		if (refltype == REFR) {

			bool into = dot(n, nl) > 0; // is ray entering or leaving refractive material?
			float nc = 1.0f;  // Index of Refraction air
			float nt = 1.5f;  // Index of Refraction glass/water
			float nnt = into ? nc / nt : nt / nc;  // IOR ratio of refractive materials
			float ddn = dot(r.dir, nl);
			float cos2t = 1.0f - nnt * nnt * (1.f - ddn * ddn);

			if (cos2t < 0.0f) // total internal reflection 
			{
				d = reflect(r.dir, n); //d = r.dir - 2.0f * n * dot(n, r.dir);
				x += nl * 0.01f;
			}
			else // cos2t > 0
			{
				// compute direction of transmission ray
				float3 tdir = normalize(r.dir * nnt - n * ((into ? 1 : -1) * (ddn * nnt + sqrtf(cos2t))));

				float R0 = (nt - nc) * (nt - nc) / (nt + nc) * (nt + nc);
				float c = 1.f - (into ? -ddn : dot(tdir, n));
				float Re = R0 + (1.f - R0) * c * c * c * c * c;
				float Tr = 1 - Re; // Transmission
				float P = .25f + .5f * Re;
				float RP = Re / P;
				float TP = Tr / (1.f - P);

				// randomly choose reflection or transmission ray
				if (hiprand_uniform(randstate) < 0.25) // reflection ray
				{
					mask *= RP;
					d = reflect(r.dir, n);
					x += nl * 0.02f;
				}
				else // transmission ray
				{
					mask *= TP;
					d = tdir; //r = Ray(x, tdir); 
					x += nl * 0.0005f; // epsilon must be small to avoid artefacts
				}
			}
		}

		// set up origin and direction of next path segment
		r.orig = x;
		r.dir = d;
	}

	// add radiance up to a certain ray depth
	// return accumulated ray colour after all bounces are computed
	return accucolor;
}

// required to convert colour to a format that OpenGL can display  
union Colour  // 4 bytes = 4 chars = 1 float
{
	float c;
	uchar4 components;
};

__global__ void render_kernel(float3* output, float3* accumbuffer, const int numscene_triangles, int framenumber, uint hashedframenumber, float3 scene_bbmin, float3 scene_bbmax) {   // float3 *gputexdata1, int *texoffsets

	// assign a CUDA thread to every pixel by using the threadIndex
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	// global threadId, see richiesams blogspot
	int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	// create random number generator, see RichieSams blogspot
	hiprandState randState; // state of the random number generator, to prevent repetition
	hiprand_init(hashedframenumber + threadId, 0, 0, &randState);

	Ray cam(firstcamorig, normalize(make_float3(0, -0.042612, -1)));
	float3 cx = make_float3(width * .5135 / height, 0.0f, 0.0f);  // ray direction offset along X-axis 
	float3 cy = normalize(cross(cx, cam.dir)) * .5135; // ray dir offset along Y-axis, .5135 is FOV angle
	float3 pixelcol; // final pixel color       

	int i = (height - y - 1) * width + x; // pixel index

	pixelcol = make_float3(0.0f, 0.0f, 0.0f); // reset to zero for every pixel	

	for (int s = 0; s < samps; s++) {

		// compute primary ray direction
		float3 d = cx * ((.25 + x) / width - .5) + cy * ((.25 + y) / height - .5) + cam.dir;
		// normalize primary ray direction
		d = normalize(d);
		// add accumulated colour from path bounces
		pixelcol += radiance(Ray(cam.orig + d * 40, d), &randState, numscene_triangles, scene_bbmin, scene_bbmax) * (1. / samps);
	}       // Camera rays are pushed ^^^^^ forward to start in interior 

	// add pixel colour to accumulation buffer (accumulates all samples) 
	accumbuffer[i] += pixelcol;
	// averaged colour: divide colour by the number of calculated frames so far
	float3 tempcol = accumbuffer[i] / framenumber;

	Colour fcolour;
	float3 colour = make_float3(clamp(tempcol.x, 0.0f, 1.0f), clamp(tempcol.y, 0.0f, 1.0f), clamp(tempcol.z, 0.0f, 1.0f));
	// convert from 96-bit to 24-bit colour + perform gamma correction
	fcolour.components = make_uchar4((unsigned char)(powf(colour.x, 1 / 2.2f) * 255), (unsigned char)(powf(colour.y, 1 / 2.2f) * 255), (unsigned char)(powf(colour.z, 1 / 2.2f) * 255), 1);
	// store pixel coordinates and pixelcolour in OpenGL readable outputbuffer
	output[i] = make_float3(x, y, fcolour.c);
}


__device__ float timer = 0.0f;

inline float clamp(float x) { return x < 0 ? 0 : x>1 ? 1 : x; }

//inline int toInt(float x){ return int(pow(clamp(x), 1 / 2.2) * 255 + .5); }  // RGB float in range [0,1] to int in range [0, 255]

// buffer for accumulating samples over several frames




// load triangle data in a CUDA texture
extern "C"
{
	void bindscene_triangles(float* dev_triangle_p, unsigned int number_of_scene_triangles)
	{
		Texture_triangle.normalized = false;                      // access with normalized texture coordinates
		Texture_triangle.filterMode = hipFilterModePoint;        // Point mode, so no 
		Texture_triangle.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(float4) * number_of_scene_triangles * 3;
		hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
		hipBindTexture(0, Texture_triangle, dev_triangle_p, channelDesc, size);
	}
}

bool firstTime = true;

void pre_render_kernel(float3* output, float3* accumbuffer, const int numscene_triangles, int framenumber, int hashedframenumber, float3 scene_bbmin, float3 scene_bbmax,
	std::vector<float4> cuda_scene_triangles,float* cudaSlabLimit,int* cudaTreeInfo, int bvh_node_num)
{

	if (firstTime)
	{
		firstTime = false;
		size_t triangle_size = cuda_scene_triangles.size() * sizeof(float4);
		int total_num_scene_triangles = cuda_scene_triangles.size() / 3;
		total_number_of_scene_triangles = total_num_scene_triangles;

		if (triangle_size > 0)
		{
			// allocate memory for the triangle meshes on the GPU
			hipMalloc((void**)&dev_triangle_p, triangle_size);

			// copy triangle data to GPU
			hipMemcpy(dev_triangle_p, &cuda_scene_triangles[0], triangle_size, hipMemcpyHostToDevice);

			// load triangle data into a CUDA texture
			bindscene_triangles(dev_triangle_p, total_num_scene_triangles);
		}
		hipChannelFormatDesc channel1desc = hipCreateChannelDesc<float2>();
		hipBindTexture(NULL, &Texture_bvh_slab, cudaSlabLimit, &channel1desc, bvh_node_num * sizeof(float2));

		hipChannelFormatDesc channel2desc = hipCreateChannelDesc<uint4>();
		hipBindTexture(NULL, &Texture_tree_info, cudaTreeInfo, &channel2desc, bvh_node_num*sizeof(uint4));


	}
	dim3 block(16, 16, 1);
	dim3 grid(width / block.x, height / block.y, 1);

	// launch CUDA path tracing kernel, pass in a hashed seed based on number of frames
	render_kernel << < grid, block >> > (dptr, accumulatebuffer, total_number_of_scene_triangles, framenumber, hashedframenumber, scene_aabbox_max, scene_aabbox_min);  // launches CUDA render kernel from the host
}



