#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#define TPB 64
//https://github.com/myurtoglu/cudaforengineers
__global__ void ddKernel(float* d_out, const float* d_in, int size, float h) {
    const int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i >= size - 1 || i == 0) return;
    d_out[i] = (d_in[i - 1] - 2.f * d_in[i] + d_in[i + 1]) / (h * h);
}

void ddParallel(float* out, const float* in, int n, float h) {
    float* d_in = 0, * d_out = 0;

    hipMalloc(&d_in, n * sizeof(float));
    hipMalloc(&d_out, n * sizeof(float));
    hipMemcpy(d_in, in, n * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t startEvent, stopEvent;
    float etime;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);

    ddKernel << <(n + TPB - 1) / TPB, TPB >> > (d_out, d_in, n, h);

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&etime, startEvent, stopEvent);
    printf(" time %f\n", etime);

    hipMemcpy(out, d_out, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_in);
    hipFree(d_out);
}


int main() {
  const float PI = 3.1415927;
  const int N = 8192;
  const float h = 2 * PI / N;
  
  float x[N] = { 0.0 };
  float u[N] = { 0.0 };
  float result_parallel[N] = { 0.0 };

  for (int i = 0; i < N; ++i) {
    x[i] = 2 * PI*i / N;
    u[i] = sinf(x[i]);
  }

  ddParallel(result_parallel, u, N, h);

  
}