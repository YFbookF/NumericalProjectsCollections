#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#define TPB 4
#define RAD 1
__global__ void ddKernel(float* d_out, const float* d_in, int size, float h) {
    const int i = threadIdx.x + blockDim.x * blockIdx.x;
    printf("blockdimx = %d,blockidxx = %d,threadidxx = %d,i = %d\n",
        blockDim.x, blockIdx.x, threadIdx.x, i);
    const int s_idx = threadIdx.x + RAD;
    extern __shared__ float s_in[];
    printf("d_in[%d] = %f\n", i - RAD, d_in[i - RAD]);
    if (threadIdx.x < RAD)
    {
        s_in[s_idx - RAD] = d_in[i - RAD];//跨线程块，d_in[-1] = 0
        s_in[s_idx + blockDim.x] = d_in[i + blockDim.x];
    }
    __syncthreads();
    d_out[i] = (s_in[s_idx - 1] - 2.f * s_in[s_idx] + s_in[s_idx + 1]) / (h * h);
}

void ddParallel(float* out, const float* in, int n, float h) {
    float* d_in = 0, * d_out = 0;

    hipMalloc(&d_in, n * sizeof(float));
    hipMalloc(&d_out, n * sizeof(float));
    hipMemcpy(d_in, in, n * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t startEvent, stopEvent;
    float etime;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);
    const size_t smemSize = (TPB + 2 * RAD) * sizeof(float);
    ddKernel << <(n + TPB - 1) / TPB, TPB ,smemSize>> > (d_out, d_in, n, h);

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&etime, startEvent, stopEvent);
    printf(" time %f\n", etime);

    hipMemcpy(out, d_out, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_in);
    hipFree(d_out);
}


int main() {
  const float PI = 3.1415927;
  const int N = 16;
  const float h = 2 * PI / N;
  
  float x[N] = { 0.0 };
  float u[N] = { 0.0 };
  float result_parallel[N] = { 0.0 };

  for (int i = 0; i < N; ++i) {
    x[i] = 2 * PI*i / N;
    u[i] = i;
  }

  ddParallel(result_parallel, u, N, h);

  
}