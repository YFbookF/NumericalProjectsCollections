#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
/*
shuffle 代替共享内存广播，N = 8192，ms = 119ms
*/
template <typename T>
__host__ __device__ void bodyBodyInteraction(
    T* fx, T* fy, T* fz,
    T x0, T y0, T z0,
    T x1, T y1, T z1, T mass1,
    T softeningSquared)
{
    T dx = x1 - x0;
    T dy = y1 - y0;
    T dz = z1 - z0;

    T distSqr = dx * dx + dy * dy + dz * dz;
    distSqr += softeningSquared;

    //
    // rsqrtf() maps to SFU instruction - to support
    // double, this has to be changed.
    //
    T invDist = rsqrtf(distSqr);

    T invDistCube = invDist * invDist * invDist;
    T s = mass1 * invDistCube;

    *fx = dx * s;
    *fy = dy * s;
    *fz = dz * s;
}


__global__ void
ComputeNBodyGravitation_Shuffle(
    float* force,
    float* posMass,
    float softeningSquared,
    size_t N)
{
    const int laneid = threadIdx.x & 31;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
        i < N;
        i += blockDim.x * gridDim.x)
    {
        float acc[3] = { 0 };
        float4 myPosMass = ((float4*)posMass)[i];

        for (int j = 0; j < N; j += 32) {
            float4 shufSrcPosMass = ((float4*)posMass)[j + laneid];
#pragma unroll 32
            for (int k = 0; k < 32; k++) {
                float fx, fy, fz;
                float4 shufDstPosMass;

                shufDstPosMass.x = __shfl(shufSrcPosMass.x, k);
                shufDstPosMass.y = __shfl(shufSrcPosMass.y, k);
                shufDstPosMass.z = __shfl(shufSrcPosMass.z, k);
                shufDstPosMass.w = __shfl(shufSrcPosMass.w, k);

                bodyBodyInteraction(
                    &fx, &fy, &fz,
                    myPosMass.x, myPosMass.y, myPosMass.z,
                    shufDstPosMass.x,
                    shufDstPosMass.y,
                    shufDstPosMass.z,
                    shufDstPosMass.w,
                    softeningSquared);
                acc[0] += fx;
                acc[1] += fy;
                acc[2] += fz;
            }
        }

        force[3 * i + 0] = acc[0];
        force[3 * i + 1] = acc[1];
        force[3 * i + 2] = acc[2];
    }
}
float
ComputeGravitation_GPU_Shuffle(
    float* force,
    float* posMass,
    float softeningSquared,
    size_t N
)
{
    hipError_t status;
    hipEvent_t evStart = 0, evStop = 0;
    float ms = 0.0;
    size_t bodiesLeft = N;

    void* p;
    hipGetSymbolAddress(&p, HIP_SYMBOL(g_constantBodies));

    hipEventCreate(&evStart);
    hipEventCreate(&evStop);
    hipEventRecord(evStart, NULL);

    ComputeNBodyGravitation_Shuffle << <300, 256 >> > (force, posMass, softeningSquared, N);

    hipEventRecord(evStop, NULL);
    hipDeviceSynchronize();
    hipEventElapsedTime(&ms, evStart, evStop);
    printf(" time %fms\n", ms);
Error:
    hipEventDestroy(evStop);
    hipEventDestroy(evStart);
    return ms;
}

int main()
{

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return -1;
    }

    int bodyNum = 8192;
    float* host_force = (float*)std::malloc(bodyNum * sizeof(float) * 4);
    float* host_posMass = (float*)std::malloc(bodyNum * sizeof(float) * 4);
    for (int i = 0; i < bodyNum * 4; i++)
    {
        host_force[i] = 0;
        host_posMass[i] = i;
    }
    float* dev_force = 0;
    float* dev_posMass = 0;
    hipMalloc((void**)&dev_force, bodyNum * 4 * sizeof(float));
    hipMalloc((void**)&dev_posMass, bodyNum * 4 * sizeof(float));
    hipMemcpy(dev_force, host_force, bodyNum * 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_posMass, host_posMass, bodyNum * 4 * sizeof(float), hipMemcpyHostToDevice);

    ComputeGravitation_GPU_Shuffle(dev_force, dev_posMass, 0.1, bodyNum);
    hipMemcpy(host_force, dev_force, bodyNum * 4 * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < bodyNum * 4; i++)
    {
        //printf("force %d = %f\n", i, host_force[i]);
    }

    hipFree(dev_force);
    hipFree(dev_posMass);
    return 0;
}

