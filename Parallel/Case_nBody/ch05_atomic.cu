#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>

/*
使用原子操作版本
N = 8192时，91ms
很难说哪个好，因为都差不多
*/
template <typename T>
__host__ __device__ void bodyBodyInteraction(
    T* fx, T* fy, T* fz,
    T x0, T y0, T z0,
    T x1, T y1, T z1, T mass1,
    T softeningSquared)
{
    T dx = x1 - x0;
    T dy = y1 - y0;
    T dz = z1 - z0;

    T distSqr = dx * dx + dy * dy + dz * dz;
    distSqr += softeningSquared;

    //
    // rsqrtf() maps to SFU instruction - to support
    // double, this has to be changed.
    //
    T invDist = rsqrtf(distSqr);

    T invDistCube = invDist * invDist * invDist;
    T s = mass1 * invDistCube;

    *fx = dx * s;
    *fy = dy * s;
    *fz = dz * s;
}


template<typename T>
__global__ void
ComputeNBodyGravitation_Atomic(T* force, T* posMass, size_t N, T softeningSquared)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
        i < N;
        i += blockDim.x * gridDim.x)
    {
        float4 me = ((float4*)posMass)[i];
        T acc[3] = { 0.0f, 0.0f, 0.0f };
        T myX = me.x;
        T myY = me.y;
        T myZ = me.z;
        for (int j = 0; j < i; j++) {
            float4 body = ((float4*)posMass)[j];

            T fx, fy, fz;
            bodyBodyInteraction(
                &fx, &fy, &fz,
                myX, myY, myZ,
                body.x, body.y, body.z, body.w,
                softeningSquared);

            acc[0] += fx;
            acc[1] += fy;
            acc[2] += fz;

            float* f = &force[3 * j + 0];
            atomicAdd(f + 0, -fx);
            atomicAdd(f + 1, -fy);
            atomicAdd(f + 2, -fz);

        }

        atomicAdd(&force[3 * i + 0], acc[0]);
        atomicAdd(&force[3 * i + 1], acc[1]);
        atomicAdd(&force[3 * i + 2], acc[2]);
    }
}
float
ComputeGravitation_GPU_Atomic(
    float* force,
    float* posMass,
    float softeningSquared,
    size_t N
)
{
    hipError_t status;
    hipEvent_t evStart = 0, evStop = 0;
    float ms = 0.0;
    size_t bodiesLeft = N;

    void* p;

    hipEventCreate(&evStart);
    hipEventCreate(&evStop);
    hipEventRecord(evStart, NULL);
    ComputeNBodyGravitation_Atomic<float> << <300, 256 >> > (force, posMass, N, softeningSquared);
    hipEventRecord(evStop, NULL);
    hipDeviceSynchronize();
    hipEventElapsedTime(&ms, evStart, evStop);
    printf(" time %fms\n", ms);
Error:
    hipEventDestroy(evStop);
    hipEventDestroy(evStart);
    return ms;
}

int main()
{

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return -1;
    }

    int bodyNum = 8192;
    float* host_force = (float*)std::malloc(bodyNum * sizeof(float) * 4);
    float* host_posMass = (float*)std::malloc(bodyNum * sizeof(float) * 4);
    for (int i = 0; i < bodyNum * 4; i++)
    {
        host_force[i] = 0;
        host_posMass[i] = i;
    }
    float* dev_force = 0;
    float* dev_posMass = 0;
    hipMalloc((void**)&dev_force, bodyNum * 4 * sizeof(float));
    hipMalloc((void**)&dev_posMass, bodyNum * 4 * sizeof(float));
    hipMemcpy(dev_force, host_force, bodyNum * 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_posMass, host_posMass, bodyNum * 4 * sizeof(float), hipMemcpyHostToDevice);

    ComputeGravitation_GPU_Atomic(dev_force, dev_posMass, 1.0, bodyNum);
    hipMemcpy(host_force, dev_force, bodyNum * 4 * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < bodyNum * 4; i++)
    {
        //printf("force %d = %f\n", i, host_force[i]);
    }
    hipFree(dev_force);
    hipFree(dev_posMass);
    return 0;
}

