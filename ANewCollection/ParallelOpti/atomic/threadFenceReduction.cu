#include "hip/hip_runtime.h"
#include <iostream>

using namespace std;

__global__ void gmem_add(int* a, int n, unsigned int* counter, int* result)
{
	bool finishSum;
	if (threadIdx.x == 0)
	{
		// 基于threadIdx.x进入延时变量
		register int  delay = blockIdx.x * 1000000;
		while (delay > 0)delay--;
		//将blockIdx.x 写入全局内存
		a[blockIdx.x] = blockIdx.x;
		__threadfence();
	}
	// 使用原子加来找到最后完成的流多处理器，记数从0开始
	if (threadIdx.x == 0)
	{
		unsigned int ticket = atomicInc(counter, gridDim.x);
		finishSum = (ticket == gridDim.x - 1);
	}
	if (finishSum)
	{
		register int sum = a[0];
#pragma unroll
		for (int i = 1; i < n; i++)sum += a[i];
		result[0] = sum;
	}
	counter = 0;
}

#define N_BLOCKS 32
int main(int argc, char* argv[])
{
	int* d_a, * d_result;
	unsigned int* d_counter;
	hipMalloc(&d_a, sizeof(int) * N_BLOCKS);
	hipMalloc(&d_result, sizeof(int));
	hipMalloc(&d_counter, sizeof(unsigned int));
	int zero = 0;
	hipMemcpy(d_counter, &zero, sizeof(int), hipMemcpyHostToDevice);
	gmem_add << <N_BLOCKS, 64 >> > (d_a, N_BLOCKS, d_counter, d_result);
	int h_a[N_BLOCKS], h_result;
	hipMemcpy(h_a, d_a, sizeof(int) * N_BLOCKS, hipMemcpyDeviceToHost);
	hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
	int sum = 0;
	for (int i = 0; i < N_BLOCKS; i++)sum += h_a[i];
	cout << "should be " << sum << " got " << h_result << endl;

}