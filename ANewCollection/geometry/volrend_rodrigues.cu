#include "hip/hip_runtime.h"
https://github.com/sxyu/volrend/blob/master/src/cuda/volrend.cu
template<typename scalar_t>
__host__ __device__ __inline__ void rodrigues(
        const scalar_t* __restrict__ aa,
        scalar_t* __restrict__ dir) {
    scalar_t angle = _norm(aa);
    if (angle < 1e-6) return;
    scalar_t k[3];
    for (int i = 0; i < 3; ++i) k[i] = aa[i] / angle;
    scalar_t cos_angle = cos(angle), sin_angle = sin(angle);
    scalar_t cross[3];
    _cross3(k, dir, cross);
    scalar_t dot = _dot3(k, dir);
    for (int i = 0; i < 3; ++i) {
        dir[i] = dir[i] * cos_angle + cross[i] * sin_angle + k[i] * dot * (1.0 - cos_angle);
    }
}
